#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include "SW_GPU.h"

#define GAP_PEN 3

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void InitNucArray (const char* RefSeq, SeqData OutSeq) {
  	int i = 1;
  	*(OutSeq.Seq + 0) = 0;
  	char CurNuc = 'B'; // Input checked by sequence input function
  	while (CurNuc != '\0')
  	{
    	CurNuc = *(RefSeq + i-1);
    	*(OutSeq.Seq + i) = (int8_t) CurNuc;
		//printf("%d ", *(OutSeq.Seq + i));
    	i++;
  	}
	printf("\n");
}

void InitSubGPMat (SeqData RefSeq, SeqData ReadSeq, int* DataMat) {
  uint32_t ReadLength = ReadSeq.SeqLength;
  uint32_t RefLength = RefSeq.SeqLength;
  int32_t i,j;
  for (i = 1; i < ReadLength; i++) {
    for (j = 1; j < RefLength; j++)
	{
      *(DataMat + i*RefLength + j) = (*(RefSeq.Seq + j) == *(ReadSeq.Seq + i)) ? MATCH_HIT : MATCH_MISS;
    }

  }
}

__global__ void kernal_scoring(SeqData *d_read, SeqData *d_ref, int * d_sub_matrix, int *d_score_matrix, int step, int ReadLength, int RefLength)
{
	int i = threadIdx.x +1; //+ blockIdx.x * blockDim.x;	
	int j = step - i;



	int16_t Score1 = 0;
	int16_t Score2 = 0;
 	int16_t Score3 = 0;	
	
	//printf("%d %d \n", i, j);	

        Score1 = *(d_score_matrix + ((i-1)*RefLength + (j-1))) + *(d_sub_matrix + (i*RefLength + j));
        Score2 = *(d_score_matrix + ((i-1)*RefLength + j)) - GAP_PEN;
        Score3 = *(d_score_matrix + (i*RefLength + (j-1))) - GAP_PEN;
	
	//printf(" %d %d %d \n ", Score1, Score2, Score3 );
	
	int16_t MaxScore = 0;
	MaxScore = (Score1 > MaxScore) ? Score1 : MaxScore;
        MaxScore = (Score2 > MaxScore) ? Score2 : MaxScore;
        MaxScore = (Score3 > MaxScore) ? Score3 : MaxScore;
	
	//printf("%d \n ", MaxScore);

	*(d_score_matrix + (i*RefLength + j)) = MaxScore;
}

int iDivUp(int a, int b){

return (a%b != 0) ? (a/b + 1) : (a/b);
}

int main(){
	//// GPU Timing variables///////
	hipEvent_t gpu_start, gpu_stop, cpu_start, cpu_stop;
	float elapsed_gpu, elapsed_cpu;
	

	///// initializing ///////////

	uint32_t RefLength = 40;  // Includes additional row needed for SW
	uint32_t ReadLength = 16; // Includes additional row needed for SW
	SeqData ReadSeq, RefSeq;
	ReadSeq.SeqLength = ReadLength;
	RefSeq.SeqLength = RefLength;

	  /* Default Sequences for testing purposes */
	  const char ref_seq_def[40] = {'C', 'A', 'G', 'C', 'C', 'T', 'T', 'T', 'C', 'T', 'G', 'A','C', 'C', 'C', 'G', 'G', 'A', 'A', 'A', 'T','C', 'A', 'A', 'A', 'A', 'T', 'A', 'G', 'G', 'C', 'A', 'C', 'A', 'A', 'C', 'A', 'A', 'A', '\0'};
	  const char read_seq_def[16] = {'C', 'T', 'G', 'A', 'G', 'C', 'C', 'G', 'G', 'T', 'A', 'A', 'A', 'T', 'C', '\0'};
	
	  /* Initialize input sequences as int8_t arrays */
	  RefSeq.Seq = (uint8_t*) malloc(RefSeq.SeqLength);	// Change to dynamic malloc depending on input
	  ReadSeq.Seq = (uint8_t*) malloc(ReadSeq.SeqLength);  // Change to dynamic malloc depending on input
	  InitNucArray (ref_seq_def, RefSeq);
	  InitNucArray (read_seq_def, ReadSeq);
	  
	  //substution matrix
 	  int* DataMat = (int*) calloc((RefLength)*(ReadLength),sizeof(int));
  	  InitSubGPMat (RefSeq, ReadSeq, DataMat);
	  
	  //scoring matrix
	  
	  int* score_matrix = (int*) calloc(((RefSeq.SeqLength)+1)*((ReadSeq.SeqLength)+1),sizeof(uint32_t));
	  
	  ////// cuda initializing ///////

	 	// Create the cuda events
       	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	// Record event  on the default stream
	hipEventRecord(gpu_start, 0);
	  
	  CUDA_SAFE_CALL(hipSetDevice(0));
	  
	  SeqData * d_read, * d_ref; 
	  int* d_score_matrix, * d_sub_matrix;
	  
	  //setting the arrays
	  CUDA_SAFE_CALL(hipMalloc((void **)&d_read, ((ReadSeq.SeqLength)*sizeof(SeqData))));
      CUDA_SAFE_CALL(hipMalloc((void **)&d_ref, ((RefSeq.SeqLength)*sizeof(SeqData))));
	  CUDA_SAFE_CALL(hipMalloc((void **)&d_score_matrix, (((RefSeq.SeqLength)+1)*((ReadSeq.SeqLength)+1)*sizeof(uint32_t))));
	  CUDA_SAFE_CALL(hipMalloc((void **)&d_sub_matrix,((RefSeq.SeqLength)*(ReadSeq.SeqLength))*sizeof(int)));
	  
	  //transfering the arrays
	  CUDA_SAFE_CALL(hipMemcpy(d_read, ReadSeq.Seq, ((ReadSeq.SeqLength)*sizeof(SeqData)), hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL(hipMemcpy(d_ref, RefSeq.Seq, ((RefSeq.SeqLength)*sizeof(SeqData)), hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL(hipMemcpy(d_sub_matrix, DataMat, ((RefSeq.SeqLength)*(ReadSeq.SeqLength))*sizeof(int), hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL(hipMemcpy(d_score_matrix, score_matrix, (((RefSeq.SeqLength)+1)*((ReadSeq.SeqLength)+1)*sizeof(uint32_t)), hipMemcpyHostToDevice ));	  

	
	  //kernal function call

	int step;
	int num_iters = RefSeq.SeqLength+ReadSeq.SeqLength-1;
	int max_thread = RefSeq.SeqLength-ReadSeq.SeqLength+1;

	int k = 0;
	//for(k = 0; k < 1600; k++){
	for(step = 1; step< num_iters; step++)
	{
		dim3 dimGrid(iDivUp(step,max_thread));

		kernal_scoring<<< dimGrid, 1>>>(d_read, d_ref, d_sub_matrix, d_score_matrix, step, ReadSeq.SeqLength, RefSeq.SeqLength);
	}
	//}
	//CUDA return
	 CUDA_SAFE_CALL(hipPeekAtLastError());
	  CUDA_SAFE_CALL(hipMemcpy(ReadSeq.Seq, d_read, ((ReadSeq.SeqLength)*sizeof(uint32_t)), hipMemcpyDeviceToHost));
	  CUDA_SAFE_CALL(hipMemcpy(RefSeq.Seq, d_ref, ((RefSeq.SeqLength)*sizeof(uint32_t)), hipMemcpyDeviceToHost));
	  CUDA_SAFE_CALL(hipMemcpy(score_matrix, d_score_matrix, (((RefSeq.SeqLength)+1)*((ReadSeq.SeqLength)+1)*sizeof(uint32_t)), hipMemcpyDeviceToHost ));	 

	

	// Stop and destroy the timer
	hipEventRecord(gpu_stop,0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&elapsed_gpu, gpu_start, gpu_stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);	

	int i,j;
//	  for (i = 0; i < RefLength; i++) {printf(" %3c ",*(RefSeq.Seq + i));}
 /* //printf("\n");
  for (i = 1; i < ReadLength; i++) {
    //printf("%c ",*(ReadSeq.Seq + i));
    for (j = 1; j < RefLength; j++) {
      printf(" %3d ", *(score_matrix + (i*RefLength + j)));
    }
	printf("\n");
	}
*/
	return 0;  

	

}
